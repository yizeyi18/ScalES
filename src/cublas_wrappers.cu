#include <cublas_wrappers.hpp>
#include <exceptions.hpp>
#include <hipblas.h>

#include "cuda_type_pimpl.hpp"

const char* cublasGetErrorString( hipblasStatus_t status ) {

  if( status == HIPBLAS_STATUS_SUCCESS )
    return "SUCCESS";
  else if( status == HIPBLAS_STATUS_NOT_INITIALIZED )
    return "NOT INITIALIZED";
  else if( status == HIPBLAS_STATUS_ALLOC_FAILED )
    return "ALLOC FAILED";
  else if( status == HIPBLAS_STATUS_INVALID_VALUE )
    return "INVALID VALUE";
  else if( status == HIPBLAS_STATUS_ARCH_MISMATCH )
    return "ARCH MISMATCH";
  else if( status == HIPBLAS_STATUS_MAPPING_ERROR )
    return "MAPPING ERROR";
  else if( status == HIPBLAS_STATUS_EXECUTION_FAILED )
    return "EXECUTION FAILED";
  else if( status == HIPBLAS_STATUS_INTERNAL_ERROR )
    return "INTERNAL ERROR";
  else if( status == HIPBLAS_STATUS_NOT_SUPPORTED )
    return "NOT SUPPORTED";
  else if( status == HIPBLAS_STATUS_UNKNOWN )
    return "INVALID LICENSE";
  else 
    return "CUBLAS ERROR NOT RECOGNIZED";
  

}

hipblasOperation_t cublasOpFromChar(char op){
	switch (op) {
		case 'n':
		case 'N':
			return HIPBLAS_OP_N;
		case 't':
		case 'T':
			return HIPBLAS_OP_T;
		case 'c':
		case 'C':
			return HIPBLAS_OP_C;
    default:
      printf("UNKNOWN CUBLAS OP - DEFAULTING TO HIPBLAS_OP_N");
      return HIPBLAS_OP_N;
	}
}






namespace cublas {
namespace detail {

  struct handle_pimpl {

    hipblasHandle_t handle;

    handle_pimpl(){
      CUBLAS_THROW( hipblasCreate( &handle ) );
    }

    ~handle_pimpl() noexcept {
      CUBLAS_ASSERT( hipblasDestroy( handle ) );
    }

  };

  handle_pimpl* get_handle( handle& h ) {
    return h.pimpl_.get();
  }

}



handle::handle() :
  pimpl_( std::make_shared<detail::handle_pimpl>() ){ }

handle::~handle() noexcept = default;
handle::handle( handle&& ) noexcept = default;
handle::handle( const handle& ) = default;

cuda::stream handle::get_stream() const {
  hipStream_t stream;
  CUBLAS_THROW( hipblasGetStream( pimpl_->handle, &stream ) );
  cuda::stream return_stream( std::make_shared<cuda::detail::stream_pimpl>() );
  return_stream.pimpl_->stream = stream;
  return return_stream;
}

void handle::set_stream( const cuda::stream& stream ) {
  CUBLAS_THROW( hipblasSetStream( pimpl_->handle, 
                                 stream.pimpl_->stream ) );
}

namespace blas {

template <>
void gemm_batched( handle& handle,
  char TRANSA, char TRANSB, int M, int N, int K, 
  double ALPHA, double** A_device, int LDA, double** B_device,
  int LDB, double BETA, double** C_device, int LDC, int batch_count ) {

  hipblasOperation_t TA = cublasOpFromChar(TRANSA) ;
  hipblasOperation_t TB = cublasOpFromChar(TRANSB) ;

  auto handle_h = detail::get_handle(handle)->handle;

  CUBLAS_THROW(
    hipblasDgemmBatched( handle_h, TA, TB, M, N, K, &ALPHA, A_device, LDA, B_device,
      LDB, &BETA, C_device, LDC, batch_count )
  )

}

template <>
void axpy( handle& handle,
  int N, double ALPHA, const double* X, int INCX, double* Y, int INCY 
) {

  auto handle_h = detail::get_handle(handle)->handle;

  CUBLAS_THROW( hipblasDaxpy( handle_h, N, &ALPHA, X, INCX, Y, INCY ) );

}

}

}
