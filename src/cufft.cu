#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipfft.h>
#include <stdlib.h>

#define NX 4
#define NY 4
int main()
{
        float *vx = (float*) malloc( NX * NY * sizeof(hipfftComplex));
        hipfftComplex *d_vx, *d_vx2;
     	int i;
    	for(i =0; i < NX *NY ; i++)
    		vx[i] = 1.0;
	assert(hipSetDevice(0) == hipSuccess);
	printf("NX NY sizoef(hipfftComplex): %d %d %d \n", NX, NY, sizeof(hipfftComplex));
        assert(hipMalloc(&d_vx,  NX*NY*sizeof(hipfftComplex)) == hipSuccess);
        assert(hipMalloc(&d_vx2, NX*NY*sizeof(hipfftComplex)) == hipSuccess);
        assert(hipMemcpy(d_vx, vx, NX*NY*sizeof(hipfftComplex), hipMemcpyHostToDevice) == hipSuccess);
    	for(i =0; i < NX *NY ; i++)
    		vx[i] = 0.0;
        hipfftHandle planr2c;
        hipfftHandle planc2r;
        (hipfftPlan2d(&planr2c, NY, NX, HIPFFT_R2C));
        (hipfftPlan2d(&planc2r, NY, NX, HIPFFT_C2R));
        //(cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE));
        //(cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE));
        assert(hipfftExecR2C(planr2c, (hipfftReal *)d_vx, d_vx2) == HIPFFT_SUCCESS);
        //hipfftExecC2R(planc2r, d_vx, (hipfftReal *)d_vx2);
        assert(hipMemcpy(vx, d_vx2, NX*NY*sizeof(hipfftComplex), hipMemcpyDeviceToHost) == hipSuccess);
	for(i =0; i < NX *NY; i++)
		printf(" vx[%d]: %f \n" , i, vx[i]);
        hipFree(d_vx);
        hipFree(d_vx2);
        free(vx);
}
