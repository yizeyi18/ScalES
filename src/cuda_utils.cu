#include "hip/hip_runtime.h"
#ifndef  _CUDA_UTILS_CU_
#define  _CUDA_UTILS_CU_
#include "cuda_utils.h"

#define DIM 128

 __device__ inline hipDoubleComplex operator* (const hipDoubleComplex & x,const hipDoubleComplex & y) {
    return hipCmul(x,y);
 }

  __device__ inline hipDoubleComplex operator+ (const hipDoubleComplex & x,const hipDoubleComplex & y) {
    return hipCadd(x,y);
 }

   __device__ inline hipDoubleComplex operator- (const hipDoubleComplex & x,const hipDoubleComplex & y) {
    return hipCsub(x,y);
 }

 __device__ inline hipDoubleComplex operator* (const double & a,const hipDoubleComplex & x) {
    return make_hipDoubleComplex (a*hipCreal(x), a*hipCimag(x));
 }

 __device__ inline hipDoubleComplex operator* (const hipDoubleComplex & x,const double & a) {
    return make_hipDoubleComplex (a*hipCreal(x), a*hipCimag(x));
 }

 __device__ inline hipDoubleComplex operator+ (const double & a,const hipDoubleComplex & x) {
    return make_hipDoubleComplex (a+hipCreal(x), hipCimag(x));
 }

 __device__ inline hipDoubleComplex operator+ (const hipDoubleComplex & x,const double & a) {
    return make_hipDoubleComplex (a+hipCreal(x), hipCimag(x));
 }
  __device__ inline double Norm_2(const hipDoubleComplex & x) {
    return (hipCreal(x)*hipCreal(x)) + (hipCimag(x)*hipCimag(x));
 }


__global__ void gpu_setValue( float* dev, float val, int len)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < len)
		dev[tid] = val;
}

__global__ void gpu_setValue( double* dev, double val, int len)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < len)
		dev[tid] = val;
}
__global__ void gpu_setValue( hipDoubleComplex* dev, hipDoubleComplex val, int len)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < len){
		dev[tid].x = val.x;
		dev[tid].y = val.y;
	}
}
__global__ void gpu_setValue( hipComplex* dev, hipComplex val, int len)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < len){
		dev[tid].x = val.x;
		dev[tid].y = val.y;
	}
}
__global__ void  gpu_interpolate_wf_C2F( hipDoubleComplex * coarse_psi, hipDoubleComplex* fine_psi, int *index, int len, double factor)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < len)
	{
		int idx = index[tid];
		fine_psi[idx] = coarse_psi[tid] * factor;
	}
}

__global__ void  gpu_interpolate_wf_F2C( hipDoubleComplex * fine_psi, hipDoubleComplex* coarse_psi, int *index, int len, double factor)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < len)
	{
		int idx = index[tid];
		coarse_psi[tid] = coarse_psi[tid] + fine_psi[idx] * factor;
	}
}
__global__ void gpu_laplacian ( hipDoubleComplex * psi, double * gkk, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < len)
	{
		psi[tid] = psi[tid] * gkk[tid];
	}
}

__global__ void gpu_vtot( double* psi, double * gkk, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < len)
	{
		psi[tid] = psi[tid] * gkk[tid];
	}
}


#if 0
float* cuda_malloc( float* ptr, size_t size)
{
  	printf("hipMalloc float the pointer %d \n", size);
	CUDA_CALL( hipMalloc( &ptr, size ) );
	return ptr;
}
double* cuda_malloc( double* ptr, size_t size)
{
  	printf("hipMalloc double the pointer %d \n", size);
	CUDA_CALL( hipMalloc( &ptr, size ) );
	return ptr;
}
hipComplex* cuda_malloc( hipComplex* ptr, size_t size)
{
  	printf("hipMalloc complex the pointer %d \n", size);
	CUDA_CALL( hipMalloc( &ptr, size ) );
	return ptr;
}
hipDoubleComplex* cuda_malloc( hipDoubleComplex* ptr, size_t size)
{
  	printf("hipMalloc double complex the pointer %d \n", size);
	CUDA_CALL( hipMalloc( &ptr, size ) );
	return ptr;
}
#endif


void cuda_free( void *ptr)
{
	CUDA_CALL( hipFree(ptr) );
}

void cuda_memcpy_CPU2GPU( void *gpu, void * cpu, size_t size )
{
	CUDA_CALL( hipMemcpy(gpu, cpu, size, hipMemcpyHostToDevice ); );
	//std::flush(std::cout);
}

void cuda_memcpy_GPU2CPU( void *cpu, void * gpu, size_t size )
{
	CUDA_CALL( hipMemcpy(cpu, gpu, size, hipMemcpyDeviceToHost); );
}

void cuda_memcpy_GPU2GPU( void * dest, void * src, size_t size)
{
	CUDA_CALL( hipMemcpy(dest, src, size, hipMemcpyDeviceToDevice); );
}

void cuda_setValue( float* dev, float val, int len )
{
	int ndim = len / DIM;
	if(len % DIM) ndim++;
	gpu_setValue<<<ndim, DIM>>>(dev, val, len);
}

void cuda_setValue( double* dev, double val, int len )
{
	int ndim = len / DIM;
	if(len % DIM) ndim++;
	gpu_setValue<<<ndim, DIM>>>(dev, val, len);
}
void cuda_setValue( hipDoubleComplex* dev, hipDoubleComplex val, int len )
{
	int ndim = len / DIM;
	if(len % DIM) ndim++;
	gpu_setValue<<<ndim, DIM>>>(dev, val, len);
}

void cuda_setValue( hipComplex* dev, hipComplex val, int len )
{
	int ndim = len / DIM;
	if(len % DIM) ndim++;
	gpu_setValue<<<ndim, DIM>>>(dev, val, len);
}

void cuda_interpolate_wf_C2F( hipDoubleComplex * coarse_psi, hipDoubleComplex * fine_psi, int * index, int len, double factor)
{
	int ndim = (len + DIM - 1) / DIM;
	gpu_interpolate_wf_C2F<<< ndim, DIM>>> ( coarse_psi, fine_psi, index, len, factor);
}
void cuda_interpolate_wf_F2C( hipDoubleComplex * fine_psi, hipDoubleComplex * coarse_psi, int * index, int len, double factor)
{
	int ndim = (len + DIM - 1) / DIM;
	gpu_interpolate_wf_F2C<<< ndim, DIM>>> ( fine_psi, coarse_psi, index, len, factor);
}

void *cuda_malloc( size_t size)
{
	void *ptr;
	CUDA_CALL( hipMalloc( &ptr, size ) );
	return ptr;
}
void cuda_laplacian( hipDoubleComplex* psi, double * gkk, int len)
{
	int ndim = (len + DIM - 1) / DIM;
	gpu_laplacian<<< ndim, DIM>>> ( psi, gkk, len);
	
}
void cuda_vtot( double* psi, double * vtot, int len)
{
	int ndim = (len + DIM - 1) / DIM;
	gpu_vtot<<< ndim, DIM>>> ( psi, vtot, len);
} 

#endif
