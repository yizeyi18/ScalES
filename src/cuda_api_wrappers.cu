#include <cuda_api_wrappers.hpp>
#include <exceptions.hpp>
#include <iostream>

namespace cuda {
namespace wrappers {

void memset( void* data, int val, size_t len ) {
  CUDA_THROW( hipMemset( data, val, len ) );
}

void device_sync() {
  CUDA_THROW( hipDeviceSynchronize() );
}

void memcpy_h2d( void* dest, const void* src, size_t len ) {
  CUDA_THROW( hipMemcpy( dest, src, len, hipMemcpyHostToDevice ) );
}

void memcpy_d2h( void* dest, const void* src, size_t len ) {
  CUDA_THROW( hipMemcpy( dest, src, len, hipMemcpyDeviceToHost ) );
}

void memcpy_d2d( void* dest, const void* src, size_t len ) {
  CUDA_THROW( hipMemcpy( dest, src, len, hipMemcpyDeviceToDevice ) );
}

void* malloc( size_t len ) {

  void* ptr;
  CUDA_THROW( hipMalloc( &ptr, len ) );
  //std::cout << "CUDA MALLOC " << len << ", " << ptr << std::endl;
  return ptr;

}

void* malloc_pinned( size_t len ) {

  void* ptr;
  CUDA_THROW( hipHostMalloc( &ptr, len ) );
  //std::cout << "CUDA MALLOC HOST" << len << ", " << ptr << std::endl;
  return ptr;

}

void* malloc_managed( size_t len ) {

  void* ptr;
  CUDA_THROW( hipMallocManaged( &ptr, len ) );
  //std::cout << "CUDA MALLOC HOST" << len << ", " << ptr << std::endl;
  return ptr;

}

void  free( void* ptr ) {
  //std::cout << "CUDA FREE " << ptr << std::endl;
  CUDA_THROW( hipFree( ptr ) );
}

void  free_pinned( void* ptr ) {
  //std::cout << "CUDA FREE Host" << ptr << std::endl;
  CUDA_THROW( hipHostFree( ptr ) );
}


}
}

