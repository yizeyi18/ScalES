#include "hip/hip_runtime.h"
#include <cuda_wrapper.hpp>
#include <iostream>
#include <cassert>
  #include <exception>

#include <hipblas.h>

//#define CUDA_THROW_AS_ASSERT

#define CUDA_ASSERT(err)  { assert( err == hipSuccess );           }
#define CUBLAS_ASSERT(err){ assert( err == HIPBLAS_STATUS_SUCCESS ); }


#ifdef CUDA_THROW_AS_ASSERT
  #define CUDA_THROW(err) CUDA_ASSERT(err);
  #define CUBLAS_THROW(err) CUBLAS_ASSERT(err);
#else
  #define CUDA_THROW(err)  { if(err != hipSuccess) throw cuda_exception( err );           }
  #define CUBLAS_THROW(err){ if(err != HIPBLAS_STATUS_SUCCESS) throw cuda_exception( err ); }
#endif

namespace cuda {
const char* cublasGetErrorString( hipblasStatus_t status ) {

  if( status == HIPBLAS_STATUS_SUCCESS )
    return "SUCCESS";
  else if( status == HIPBLAS_STATUS_NOT_INITIALIZED )
    return "NOT INITIALIZED";
  else if( status == HIPBLAS_STATUS_ALLOC_FAILED )
    return "ALLOC FAILED";
  else if( status == HIPBLAS_STATUS_INVALID_VALUE )
    return "INVALID VALUE";
  else if( status == HIPBLAS_STATUS_ARCH_MISMATCH )
    return "ARCH MISMATCH";
  else if( status == HIPBLAS_STATUS_MAPPING_ERROR )
    return "MAPPING ERROR";
  else if( status == HIPBLAS_STATUS_EXECUTION_FAILED )
    return "EXECUTION FAILED";
  else if( status == HIPBLAS_STATUS_INTERNAL_ERROR )
    return "INTERNAL ERROR";
  else if( status == HIPBLAS_STATUS_NOT_SUPPORTED )
    return "NOT SUPPORTED";
  else if( status == HIPBLAS_STATUS_UNKNOWN )
    return "INVALID LICENSE";
  else 
    return "CUBLAS ERROR NOT RECOGNIZED";
  

}

hipblasOperation_t cublasOpFromChar(char op){
	switch (op) {
		case 'n':
		case 'N':
			return HIPBLAS_OP_N;
		case 't':
		case 'T':
			return HIPBLAS_OP_T;
		case 'c':
		case 'C':
			return HIPBLAS_OP_C;
    default:
      printf("UNKNOWN CUBLAS OP - DEFAULTING TO HIPBLAS_OP_N");
      return HIPBLAS_OP_N;
	}
}

class cuda_exception : public std::exception {

  std::string message;

  virtual const char* what() const throw() {
    return message.c_str();
  }

public:

  cuda_exception( const char* msg ) : std::exception(), message( msg ) { };
  cuda_exception( hipError_t err ) : cuda_exception( hipGetErrorString( err ) ) { } 
  cuda_exception( hipblasStatus_t err ) : cuda_exception( cublasGetErrorString( err ) ) { }

};

namespace wrappers {

void memset( void* data, int val, size_t len ) {
  CUDA_THROW( hipMemset( data, val, len ) );
}

void device_sync() {
  CUDA_THROW( hipDeviceSynchronize() );
}

void memcpy_h2d( void* dest, const void* src, size_t len ) {
  CUDA_THROW( hipMemcpy( dest, src, len, hipMemcpyHostToDevice ) );
}

void memcpy_d2h( void* dest, const void* src, size_t len ) {
  CUDA_THROW( hipMemcpy( dest, src, len, hipMemcpyDeviceToHost ) );
}

void* malloc( size_t len ) {

  void* ptr;
  CUDA_THROW( hipMalloc( &ptr, len ) );
  //std::cout << "CUDA MALLOC " << len << ", " << ptr << std::endl;
  return ptr;

}

void  free( void* ptr ) {
  //std::cout << "CUDA FREE " << ptr << std::endl;
  CUDA_THROW( hipFree( ptr ) );
}

}


namespace detail {

  struct cuda_event_pimpl {

    hipEvent_t event;

    cuda_event_pimpl(){
      CUDA_THROW( hipEventCreate( &event ) );
    }

    ~cuda_event_pimpl() noexcept {
      CUDA_ASSERT( hipEventDestroy( event ) );
    }

  };

  struct cuda_stream_pimpl {

    hipStream_t stream;

    cuda_stream_pimpl(){
      CUDA_THROW( hipStreamCreate( &stream ) );
    }

    ~cuda_stream_pimpl() noexcept {
      CUDA_ASSERT( hipStreamDestroy( stream ) );
    }

  };

  struct cublas_handle_pimpl {

    hipblasHandle_t handle;

    cublas_handle_pimpl(){
      CUBLAS_THROW( hipblasCreate( &handle ) );
    }

    ~cublas_handle_pimpl() noexcept {
      CUBLAS_ASSERT( hipblasDestroy( handle ) );
    }

  };

}


hip_stream::hip_stream() :
  pimpl_( std::make_unique<detail::cuda_stream_pimpl>() ){ }

hip_stream::~hip_stream() noexcept = default;
hip_stream::hip_stream( hip_stream&& ) noexcept = default;

void hip_stream::synchronize() const {
  CUDA_THROW( hipStreamSynchronize( pimpl_->stream ) );
}






cuda_event::cuda_event() :
  pimpl_( std::make_unique<detail::cuda_event_pimpl>() ){ }

cuda_event::~cuda_event() noexcept = default;
cuda_event::cuda_event( cuda_event&& ) noexcept = default;

void cuda_event::record( const hip_stream& stream ) {
  CUDA_THROW( hipEventRecord( pimpl_->event, stream.pimpl_->stream ) );
}

void cuda_event::record() {
  CUDA_THROW( hipEventRecord( pimpl_->event ) );
}

void cuda_event::synchronize() const {
  CUDA_THROW( hipEventSynchronize( pimpl_->event ) );
}


float cuda_event::elapsed_time( const cuda_event& first, const cuda_event& second ) {
  float time;
  CUDA_THROW( hipEventElapsedTime( &time, first.pimpl_->event, second.pimpl_->event ) );
  return time;
}

cublas_handle::cublas_handle() :
  pimpl_( std::make_unique<detail::cublas_handle_pimpl>() ){ }

cublas_handle::~cublas_handle() noexcept = default;
cublas_handle::cublas_handle( cublas_handle&& ) noexcept = default;

detail::cublas_handle_pimpl* cublas_handle::pimpl() const{
  return pimpl_.get();
};








template <>
void cublas_gemm_batched( cublas_handle& handle,
  char TRANSA, char TRANSB, int M, int N, int K, 
  double ALPHA, double** A_device, int LDA, double** B_device,
  int LDB, double BETA, double** C_device, int LDC, int batch_count ) {

  hipblasOperation_t TA = cublasOpFromChar(TRANSA) ;
  hipblasOperation_t TB = cublasOpFromChar(TRANSB) ;

  auto handle_h = handle.pimpl()->handle;

  CUBLAS_THROW(
    hipblasDgemmBatched( handle_h, TA, TB, M, N, K, &ALPHA, A_device, LDA, B_device,
      LDB, &BETA, C_device, LDC, batch_count )
  )

}

template <>
void cublas_axpy( cublas_handle& handle,
  int N, double ALPHA, const double* X, int INCX, double* Y, int INCY 
) {

  auto handle_h = handle.pimpl()->handle;

  CUBLAS_THROW( hipblasDaxpy( handle_h, N, &ALPHA, X, INCX, Y, INCY ) );

}

template <typename T>
__global__ void axpby_kernel( const size_t n, const T alpha, const T* X, const size_t INCX, const T beta, T* Y, const size_t INCY ) {

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid < n ) {
		const int tid_inc_y = tid * INCY;
		const int tid_inc_x = tid * INCX;
		Y[ tid_inc_y ] = beta * Y[ tid_inc_y ] + alpha * X[ tid_inc_x ];
	}
}


template <typename T>
void axpby_device( 
  int N, T ALPHA, const T* X, int INCX, T BETA, T* Y, int INCY
) {

  auto div = std::div( N, 1024 );
  
  axpby_kernel<T><<< div.quot + !!div.rem, 1024 >>>( 
    N, ALPHA, X, INCX, BETA, Y, INCY 
  );
  CUDA_THROW( hipGetLastError() );
}

template
void axpby_device<double>( 
  int N, double ALPHA, const double* X, int INCX, double BETA, double* Y, int INCY
);

}
