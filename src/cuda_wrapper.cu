#include "hip/hip_runtime.h"
#include <cuda_wrapper.hpp>
#include <cassert>
#include <iostream>

#include <hipblas.h>

namespace cuda {
namespace wrappers {

void memset( void* data, int val, size_t len ) {
  auto status = hipMemset( data, val, len );
}

void device_sync() {
  hipDeviceSynchronize();
}

void memcpy_h2d( void* dest, const void* src, size_t len ) {
  auto status = hipMemcpy( dest, src, len, hipMemcpyHostToDevice );
  assert( status == hipSuccess );
}

void memcpy_d2h( void* dest, const void* src, size_t len ) {
  auto status = hipMemcpy( dest, src, len, hipMemcpyDeviceToHost );
  assert( status == hipSuccess );
}

void* malloc( size_t len ) {

  void* ptr;
  auto status =hipMalloc( &ptr, len );
  assert( status == hipSuccess );
  //std::cout << "CUDA MALLOC " << len << ", " << ptr << std::endl;
  return ptr;

}

void  free( void* ptr ) {
  //std::cout << "CUDA FREE " << ptr << std::endl;
  auto status = hipFree( ptr );
  assert( status == hipSuccess );
}

}


namespace detail {

  struct cuda_event_pimpl {

    hipEvent_t event;

    cuda_event_pimpl(){
      hipEventCreate( &event );
    }

    ~cuda_event_pimpl() noexcept {
      hipEventDestroy( event );
    }

  };

  struct cuda_stream_pimpl {

    hipStream_t stream;

    cuda_stream_pimpl(){
      hipStreamCreate( &stream );
    }

    ~cuda_stream_pimpl() noexcept {
      hipStreamDestroy( stream );
    }

  };

  struct cublas_handle_pimpl {

    hipblasHandle_t handle;

    cublas_handle_pimpl(){
      hipblasCreate( &handle );
    }

    ~cublas_handle_pimpl() noexcept {
      hipblasDestroy( handle );
    }

  };

}


hip_stream::hip_stream() :
  pimpl_( std::make_unique<detail::cuda_stream_pimpl>() ){ }

hip_stream::~hip_stream() noexcept = default;
hip_stream::hip_stream( hip_stream&& ) noexcept = default;

void hip_stream::synchronize() const {
  auto status = hipStreamSynchronize( pimpl_->stream );
  assert( status == hipSuccess );
}






cuda_event::cuda_event() :
  pimpl_( std::make_unique<detail::cuda_event_pimpl>() ){ }

cuda_event::~cuda_event() noexcept = default;
cuda_event::cuda_event( cuda_event&& ) noexcept = default;

void cuda_event::record( const hip_stream& stream ) {
  auto status = hipEventRecord( pimpl_->event, stream.pimpl_->stream );
  assert( status == hipSuccess );
}

void cuda_event::record() {
  auto status = hipEventRecord( pimpl_->event );
  assert( status == hipSuccess );
}

void cuda_event::synchronize() const {
  auto status = hipEventSynchronize( pimpl_->event );
  assert( status == hipSuccess );
}


float cuda_event::elapsed_time( const cuda_event& first, const cuda_event& second ) {
  float time;
  auto status = hipEventElapsedTime( &time, 
    first.pimpl_->event, second.pimpl_->event );
  assert( status == hipSuccess );
  return time;
}

cublas_handle::cublas_handle() :
  pimpl_( std::make_unique<detail::cublas_handle_pimpl>() ){ }

cublas_handle::~cublas_handle() noexcept = default;
cublas_handle::cublas_handle( cublas_handle&& ) noexcept = default;

detail::cublas_handle_pimpl* cublas_handle::pimpl() const{
  return pimpl_.get();
};








template <>
void cublas_gemm_batched( cublas_handle& handle,
  char TRANSA, char TRANSB, int M, int N, int K, 
  double ALPHA, double** A_device, int LDA, double** B_device,
  int LDB, double BETA, double** C_device, int LDC, int batch_count ) {

  hipblasOperation_t TA = TRANSA == 'N' ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t TB = TRANSB == 'N' ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  auto handle_h = handle.pimpl()->handle;

  auto stat = 
  hipblasDgemmBatched( handle_h, TA, TB, M, N, K, &ALPHA, A_device, LDA, B_device,
    LDB, &BETA, C_device, LDC, batch_count );

  assert( stat == HIPBLAS_STATUS_SUCCESS );

}

template <>
void cublas_axpy( cublas_handle& handle,
  int N, double ALPHA, const double* X, int INCX, double* Y, int INCY 
) {

  auto handle_h = handle.pimpl()->handle;

  auto stat = 
    hipblasDaxpy( handle_h, N, &ALPHA, X, INCX, Y, INCY );

  assert( stat == HIPBLAS_STATUS_SUCCESS );
}

template <typename T>
__global__ void axpby_kernel( const size_t n, const T alpha, const T* X, const size_t INCX, const T beta, T* Y, const size_t INCY ) {

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid < n ) {
		const int tid_inc_y = tid * INCY;
		const int tid_inc_x = tid * INCX;
		Y[ tid_inc_y ] = beta * Y[ tid_inc_y ] + alpha * X[ tid_inc_x ];
	}
}


template <typename T>
void axpby_device( 
  int N, T ALPHA, const T* X, int INCX, T BETA, T* Y, int INCY
) {
  axpby_kernel<T><<< std::ceil( N / 1024.0 ), 1024 >>>( 
    N, ALPHA, X, INCX, BETA, Y, INCY 
  );
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
    printf( "AXPBY Error!: %s\n", hipGetErrorString( err ) );
}

template
void axpby_device<double>( 
  int N, double ALPHA, const double* X, int INCX, double BETA, double* Y, int INCY
);

}
